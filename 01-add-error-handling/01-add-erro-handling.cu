
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;
  hipError_t err;
  
  size_t size = N * sizeof(int);
  err = hipMallocManaged(&a, size);
  if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));
      
  init(a, N);

  int threads_per_block = 2048;  // INVALID
  int number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  err = hipGetLastError();
  if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));
  
  err = hipDeviceSynchronize();
  if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  err = hipFree(a);
  if (err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(err));
}
